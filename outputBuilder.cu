#include "hip/hip_runtime.h"
#include "outputBuilder.h"

// Takes in a list of mpz_t bad RSA moduli and prints them alongside their private keys
// Output format per moduli/line: "<moduli>:<private key>\n"
// Makes the questionably dangerous assumptiong that badModuliFlags is always 32 uints long.
void printOutput (mpz_t *moduli, uint32_t *badModuliFlags, int totalModuliCount) {

    // badModuliFlags contains a packed set of bit flags that represent if a particular moduli is bad
    // We must first serially re-compute the GCD of the bad moduli and when we do, print them alongside their private keys

    // Generate new array of JUST BAD moduli
    mpz_t *badModuli = (mpz_t*) malloc(totalModuliCount * sizeof(mpz_t));
    int badModuliCount = 0;
    uint32_t bitMask = 0;
    for (int i = 0; i < ceil(totalModuliCount/(1.0*UINT32_LENGTH)); ++i)
    {
        bitMask = 0x00000001; bitMask <<= 31;
        for (int bitCtr = 0; bitCtr < UINT32_LENGTH; ++bitCtr)
        {
            if((badModuliFlags[i] & bitMask) != 0) {
		    mpz_init(badModuli[badModuliCount]);
		    mpz_set(badModuli[badModuliCount++], moduli[i*32 + bitCtr]);
            }
            bitMask >>= 1;
        }
    }

    mpz_t gcd;
    mpz_init(gcd);

    // Recompute the GCDs serially
    for (int i = 0; i < badModuliCount; i++)
    {
        for (int j = (i+1); j < badModuliCount; j++)
        {
            mpz_clear(gcd); mpz_init(gcd);
	    mpz_gcd(gcd, badModuli[i], badModuli[j]);
            if(mpz_cmp_ui(gcd, 1) > 0) {

                // calculate private key
                mpz_t privateKey;
                mpz_init(privateKey);
                generatePrivateKeyFromModulusAndPrime(privateKey, badModuli[i], gcd);
                gmp_printf("%Zd:%Zd\n", badModuli[i], privateKey);
                generatePrivateKeyFromModulusAndPrime(privateKey, badModuli[j], gcd);
                gmp_printf("%Zd:%Zd\n", badModuli[j], privateKey);
            }
        }
    }
}

// void generatePrivateKeyFromModulusAndPrime(mpz_t privateKey, mpz_t modulus, mpz_t prime) {
    
// }

void generatePrivateKeyFromModulusAndPrime(mpz_t privateKey, mpz_t modulus, mpz_t prime) {

    mpz_t p; mpz_init(p); mpz_set(p, prime);
    mpz_t q; mpz_init(q); mpz_div(q, modulus, p);
    mpz_t n; mpz_init(n); mpz_mul(n, p, q);

    mpz_sub_ui(p, p, 1);
    mpz_sub_ui(q, q, 1);
    mpz_t e;       mpz_init(e);       mpz_set_ui(e, 65537);
    mpz_t totient; mpz_init(totient); mpz_mul(totient, p, q);
   
    // Calculate d
    mpz_t negativeOne;
    mpz_init(negativeOne);
    mpz_set_si(negativeOne, -1);
    mpz_t d; mpz_init(d);
    mpz_powm(d, e, negativeOne, totient);

    mpz_set(privateKey, d); 
}
