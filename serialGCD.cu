#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/stat.h>
#include <gmp.h>
#include "outputBuilder.h"

#define MODULI_BUF_SIZE 2000

int main(int argc, char const *argv[])
{

    if(argc < 2) {
        fprintf(stderr, "Usage: %s <file name>\n", argv[0]);
        return -1;
    }

    // read in file/moduli
    // -------------------------------------------------------------------------
    FILE *fp = fopen((char *)argv[1], "r");

    if (fp && !feof(fp)) {

        int numModuli;

        // TODO: Is this bad? Should we be mallocing?
        mpz_t *moduli = (mpz_t*) malloc(MODULI_BUF_SIZE * sizeof(mpz_t));
        int i = 0, scanResult, moduliArraySize = MODULI_BUF_SIZE;

        do {
            mpz_init(moduli[i]);
            scanResult = gmp_fscanf(fp, "%Zd", moduli[i++]);
            // Resize moduli array if necessary
            if(i == moduliArraySize) {
                // TODO: You best not read in over 2.14 billion elements
                moduliArraySize *= 2;
                moduli = (mpz_t*) realloc(moduli, (moduliArraySize * sizeof(mpz_t)));
            }
        } while(scanResult > 0);

        numModuli = i-1;

        // GCD all pairs
        // ---------------------------------------------------------------------

        mpz_t gcd;
        mpz_init(gcd);

        for (i = 0; i < numModuli; i++)
        {
            int j;
            for (j = (i+1); j < numModuli; j++)
            {
                mpz_clear(gcd); mpz_init(gcd);
                mpz_gcd(gcd, moduli[i], moduli[j]);

                if(mpz_cmp_ui(gcd, 1) > 0) {
                    // If it's a bad key, print its stuff
                    mpz_t privateKey; mpz_init(privateKey);
                    generatePrivateKeyFromModulusAndPrime(privateKey, moduli[i], gcd);
                    gmp_printf("%Zd:%Zd\n", moduli[i], privateKey);

                    mpz_clear(privateKey); mpz_init(privateKey);
                    generatePrivateKeyFromModulusAndPrime(privateKey, moduli[j], gcd);
                    gmp_printf("%Zd:%Zd\n", moduli[j], privateKey);
                }

            }
        }

        fclose(fp);
    }
    else {
        fprintf(stderr, "%s, doesn\'t appear to be a proper file.\n", argv[1]);
        fprintf(stderr, "Are you sure it\'s a file of moduli?\n");
        return -1;
    }
}

void euclidianGCD(mpz_t *gcd, mpz_t numA, mpz_t numB) {
 
    mpz_t quotient; mpz_init(quotient);
    mpz_t remainder; mpz_init(remainder);

    // if either number is <= 0, bail
    if (mpz_cmp_ui(numA, 0) <= 0 || mpz_cmp_ui(numB, 0) <= 0) {
        fprintf(stderr, "Cannot calculate GCD of non positive integer. GCD has not been set.\n");       
        return;
    }

    // if A is smaller than B then swap them
    if (mpz_cmp(numA, numB) < 0) {
        mpz_t temp;
        mpz_set(temp, numA);
        mpz_set(numA, numB);
        mpz_set(numB, temp);
    }

    do {
        mpz_tdiv_qr(quotient, remainder, numA, numB);

        if(mpz_cmp_ui(remainder, 0) == 0) {
            mpz_set(*gcd, numB);
            return;            
        }

        else {
            mpz_set(numA, numB);
            mpz_set(numB, remainder);
        }

    } while (mpz_cmp_ui(remainder, 0) > 0); // Should this just be while(true)
}
