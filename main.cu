#include "bigInt.c"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cudaFunctions.h"

#define MODULI_BUF_SIZE 2000

int main(int argc, char const *argv[])
{
    if(argc < 2) {
        fprintf(stderr, "Usage: %s <file name>\n", argv[0]);
        return -1;
    }

    bigInt **moduli = (bigInt**) malloc(MODULI_BUF_SIZE * sizeof(bigInt**));

    // Read in the bigInts
    int numModuli = readBigIntsFromFile(argv[1], moduli);

    if(numModuli < 0) {
        fprintf(stderr, "No moduli read. Exiting.\n");
        return -1;
    }

    printf("%d moduli read in.\n", numModuli);

    // send array to CUDA
    // -------------------------------------------------------------------------
    
    // int *keys;
    // hipMalloc(keys, numModuli*sizeof(bigInt));
    // hipMemcpy(keys, moduli, numModuli*sizeof(bigInt), cudaMemCpyDeviceToHost);

    // get back bit array
    // -------------------------------------------------------------------------

    // calculate and print results
    // -------------------------------------------------------------------------

    return 0;
}
