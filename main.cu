#include "hip/hip_runtime.h"
#include "bigInt.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cudaFunctions.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>

#define MODULI_BUF_SIZE 2000
#define MAX_LENGTH_OF_1024_BIT_NUM 311

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/* This function opens a file, and maps all of its contents
 * to memory. It returns the pointer to this mapping.
 */
char *getFileContents(char* filename, size_t *len) {
   char *fileMapping;
   int file = open(filename, O_RDONLY);
   struct stat filestats;
   
   /*Check if file exists to grab contents*/
   if (file < 0) {
      printf("Error: %s does not exist or can't be read\n", filename);
      exit(EXIT_FAILURE);
   }
   
   /*Make sure that the stats retrieved by the file are valid*/
   if(fstat(file, &filestats) < 0) {
      printf("Error: stats could not be read by %s\n", filename);
      exit(EXIT_FAILURE);
   }
   
   /*Create the file mapping and return the result*/
   fileMapping = (char *) mmap(0, filestats.st_size, PROT_READ|PROT_WRITE, MAP_PRIVATE, file, 0);
   *len = filestats.st_size;
   
   if(close(file) < 0) {
      printf("Error: could not close %s after mapping.\n", filename);
      exit(EXIT_FAILURE);
   }
   
   return fileMapping;

}

/* This function goes through a mapping of a file
 * to compute the number of rows and columns of
 * the matrix represented in the mapping.
 */
void getFileStats(char *file, int *numKeys) {
   char *saved;
   /*Newlines represent rows*/
   char* sep1 = "\n";
   *numKeys = 0;

   /*Parse the mapping until all rows are counted*/
   strtok_r(file, sep1, &saved);
   (*numKeys)++;

   while(strtok_r(NULL, sep1, &saved)) {
      (*numKeys)++;
   }
}


int main(int argc, char *argv[])
{
   bigInt *moduli;
   bigInt newBigInt;
   bigInt *cuModuli;
   //dim3 dimBlock(32, 1);
   dim3 dimGrid(1);
   char* file;
   FILE *fp;
   uint32_t *bitVec;
   uint32_t *cuBitVec;
   size_t len;
   int i = 0, j;
   int numKeys = 0;
   uint32_t mask;
   int count = 0;
   dim3 dimBlock(32, 1);
   bigInt *num1;
   bigInt *num2;
   bigInt *cuNum1;
   bigInt *cuNum2;
   bigInt *cuSubRes;
   bigInt *subRes;

   if(argc < 2) {
        fprintf(stderr, "Usage: %s <file name>\n", argv[0]);
        return -1;
    }

    file = getFileContents(argv[1], &len);
      
    /* Check the validity of the mappings from both
     * files.
     */
    if (file == MAP_FAILED) {
      printf("Error: invalid map to one or more files\n");
      exit(EXIT_FAILURE);
    }
    getFileStats(file, &numKeys);
    munmap(file, len);
    printf("numkeys: %d\n", numKeys);
    moduli = (bigInt*) malloc(numKeys*sizeof(bigInt));
    bitVec = (uint32_t*) malloc(ceil(numKeys/32.0)*sizeof(uint32_t));

    bigInt *moduli = (bigInt*) malloc(MODULI_BUF_SIZE * sizeof(bigInt**));

    // Read in the bigInts
    int numModuli = readBigIntsFromFile(argv[1], moduli);

    if(numModuli < 0) {
        fprintf(stderr, "No moduli read. Exiting.\n");
        return -1;
    }

    printf("%d moduli read in.\n", numModuli);

    // send array to CUDA
    // -------------------------------------------------------------------------
    /*hipMalloc((void **) &cuModuli, numKeys*sizeof(bigInt));
    hipMemcpy((void *) cuModuli, (void *) moduli, numKeys*sizeof(bigInt), hipMemcpyHostToDevice);
    hipMalloc((void **) &cuBitVec, ceil(numKeys/32.0)*sizeof(uint32_t));
    hipMemset((void *) cuBitVec, 0, ceil(numKeys/32.0)*sizeof(uint32_t)); */ 
    /*//for(i = 0; i < numKeys; i++) {
       //for(j = i + 1; j < numKeys; j++) {
         gcdKernel<<<dimGrid, dimBlock>>>(64, 66, cuModuli, numKeys, cuBitVec);
      //}
    //}
    hipMemcpy((void *) bitVec, (void *) cuBitVec, ceil(numKeys/32.0)*sizeof(uint32_t), hipMemcpyDeviceToHost);
    
    for(i = 0; i < ceil(numKeys/32.0); i++) {
       mask = 1;
       for(j = 0; j < 32; j++) {
          if(bitVec[i] & (mask << j)) {
             count ++;
          }
       }
    }
    printf("numBadKeys: %d\n", count);*/

    //num1.components = (uint32_t *) malloc(32*sizeof(uint32_t));
    //num2.components = (uint32_t *) malloc(32*sizeof(uint32_t));
    //subRes.components = (uint32_t *) malloc(32*sizeof(uint32_t));
    //for(i = 0; i < 32; i++) {
      // subRes.components[i] = 0;
    //}
    num1 = (bigInt *) malloc(sizeof(bigInt));
    num2 = (bigInt *) malloc(sizeof(bigInt));
    subRes = (bigInt *) malloc(sizeof(bigInt));
    num1->components[0] = 30;
    num2->components[0] = 5;

    for(i = 1; i < 32; i++) {
       num1->components[i] = 0;
       num2->components[i] = 0;
    }
    //num1.components[31] = 45;
    //num2.components[31] = 44;
    //num1.components[1] = 33;
    //num2.components[1] = 33;
    //num1.components[31] = 45;
    //num2.components[31] = 45;
    //num1.components[30] = 0;
    //num1.components[29] = 0;
    //num2.components[0] = ;
    hipMalloc((void **) &cuNum1, sizeof(bigInt));
    hipMalloc((void **) &cuNum2, sizeof(bigInt));
    hipMalloc((void **) &cuSubRes, sizeof(bigInt));
    //hipMemset((void *) cuSubRes.components, 0, 32*sizeof(uint32_t));
    hipMemcpy((void *) cuNum1, (void *) num1, sizeof(bigInt), hipMemcpyHostToDevice);
    hipMemcpy((void *) cuNum2, (void *) num2, sizeof(bigInt), hipMemcpyHostToDevice);
    subTest<<<1, dimBlock>>>(cuNum1, cuNum2, cuSubRes);
    HANDLE_ERROR(hipMemcpy((void *) subRes, (void *) cuSubRes, sizeof(bigInt), hipMemcpyDeviceToHost));

    for(i = 31; i >=0; i--) {
       printf("%u ", subRes->components[i]);
    }
    printf("\n");

    // get back bit array
    // -------------------------------------------------------------------------

    // calculate and print results
    // -------------------------------------------------------------------------

    return 0;
}
