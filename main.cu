#include "bigInt.c"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define MODULI_BUF_SIZE 2000
#define MAX_LENGTH_OF_1024_BIT_NUM 311

int main(int argc, char const *argv[])
{
    if(argc < 2) {
        fprintf(stderr, "Usage: %s <file name>\n", argv[0]);
        return -1;
    }

    bigInt **moduli = (bigInt**) malloc(MODULI_BUF_SIZE * sizeof(bigInt**));

    // read in file
    // -------------------------------------------------------------------------
    FILE *fp = fopen((char *)argv[1], "r");


    // create array of bigInts
    // -------------------------------------------------------------------------
    int i = 0;
    int moduliArraySize = MODULI_BUF_SIZE;

    if (fp && !feof(fp)) {

        char bigIntString[MAX_LENGTH_OF_1024_BIT_NUM] = "\0";

        while (fgets(bigIntString, MAX_LENGTH_OF_1024_BIT_NUM, fp)) {
          
            // Trim new line
            bigIntString[strlen(bigIntString) - 1] = '\0';

            // Create bigInt
            bigInt *newBigInt = (bigInt*) malloc(sizeof(bigInt));
            initBigInt(newBigInt);
            setBigIntFromString(newBigInt, bigIntString);

            // Resize moduli array if necessary
            if(i == moduliArraySize) {
                // TODO: You best not read in over 2.14 billion elements
                moduliArraySize *= 2;
                moduli = (bigInt**) realloc(moduli, (moduliArraySize * sizeof(bigInt**)));
            }

            // Assign into moduli array
            moduli[i++] = newBigInt;
        }
    }

    fclose(fp);

    printf("%d moduli read in.\n", i);

    // send array to CUDA
    // -------------------------------------------------------------------------

    // get back bit array
    // -------------------------------------------------------------------------

    // calculate and print results
    // -------------------------------------------------------------------------

    return 0;
}
