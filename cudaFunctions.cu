#include "hip/hip_runtime.h"
#include "cudaFunctions.h"

__global__ void gcdKernel(int base, int offset, BigInt keys[], int numKeys, char results[]) {
   
   int key1 = base + threadIdx.y;
   int key2 = offset + blockIdx.x;

   __shared__ BigInt sharedkeys[BLOCKDIM_Y][2];
   BigInt *res;
   
   //__shared__ int results[BLOCKDIM_Y][BLOCKDIM_X];
   
   if(key1 < key2 && key2 < numKeys && key1 < numKeys) {
      sharedkeys[threadIdx.y][1] = keys[key1];
      sharedkeys[threadIdx.y][2] = keys[key2];
      res = gcd(&sharedkeys[threadIdx.y][1], &sharedkeys[threadIdx.y][2]);
      if(notOne(*res)) {
         results[key1*gridDim.x + key2] = 1;
      }
   }
   //BigInt n = keys[ind1+row];
   //BigInt tmp = n;
   //BigInt m; 
   
   /*for(i = ind1+row + 1; i < totNumKeys; i++) {
      m = keys[i];
      m = gcd(tmp, m);
      if(notOne(m)) {
         res[totNumKeys*(ind1+row) + i] = 1;
      }

   }*/
}

__device__ void  shiftR(BigInt *n) {
   unsigned int part = 0;
   int id = threadIdx.x;

   if(id != 31) {
      part = n.components[threadIdx.x + 1];
   }
   n.components[threadIdx.x] = (n.components[threadIdx.x] >> 1) | (part << 31);
   
   /*unsigned int part = n->components[threadIdx.x];
   __shared__ unsigned int rsResult[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   unsigned int mask = 1;

   if(threadIdx.x != 0) {
      rsResult[threadIdx.y][threadIdx.x - 1] |= (part & mask) << 31;
   }

   part >>= 1;

   if(threadIdx.x != blockDim.x - 1) {
      rsResult[threadIdx.y][threadIdx.x] |= part;
   }
   __syncThreads();
   n->components[threadIdx.x] = rsResult[threadIdx.y][threadIdx.x];
   __syncthreads();*/
}

__device__ void  shiftL(BigInt *n) {
   unsigned int part = 0;

   if(ThreadIdx.x) {
      part = n.components[threadIdx.x - 1];
   }
   n.components[threadIdx.x] = (n.components[threadIdx.x] << 1) | (part >> 31);

   /*unsigned int part = n->components[threadIdx.x];
   __shared__ unsigned int lOverflow[BLOCKDIM_Y][BLOCKDIM_X];
   unsigned int mask = 1 << 31;

   if(threadIdx.x != blockDim.x - 1) {
      lOverflow.components[threadIdx.y][threadIdx.x - 1] = (part & mask) >> 31;
   }

   part <<= 1;
   __syncthreads();

   if(threadIdx.x != 0)
      part = part | lOverflow.components[threadIdx.y][threadIdx.x];
   }

   n->components[threadIdx.x] = part;
   __syncthreads();*/
}

__device__ void cuSubtract(BigInt *n, BigInt *m) {

   unsigned int partn = n->components[threadIdx.x];
   unsigned int partm = m->components[threadIdx.x];
   __shared__ unsigned int borrowArray[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   //__shared__ short okayToGo[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   //unsigned int result = 0;
   unsigned int tmp;

   //partm = ~partm;

   /*if(threadIdx.x == 31) {
      borrowArray[threadIdx.y][0] = 0;
   }*/

   tmp = partn - partm;
   if(threadIdx != 0 && tmp > partn) {
      borrowArray[threadIdx.y][threadIdx.x + 1] = 1;
   }
   else {
      borrowArray[threadIdx.y][threadIdx.x] = 0;
   }

   while(__any(borrowArray[threadIdx.y][threadIdx.x])) {
      if(borrowArray[threadIdx.y][threadIdx.x]) {
         tmp --;
      }
      if(threadIdx.x != 31 && t == 0xffffffffU) {
         borrowArray[threadIdx.y][threadIdx.x + 1] = 1;
      }
      else {
         borrowArray[threadIdx.y][threadIdx.x + 1] = 0;
      }
   }
   n->components[threadIdx.x] = tmp;



      /*tmp = partm++;
      if(tmp < partm) {
         carry[threadIdx.y][threadIdx.x + 1] += 1;
      }
      partm = tmp;*/
   /*}

   res = partn + partm;
   if((res < partn || res < partm) && threadIdx.x < BLOCKDIM - 1) {
      carry[threadIdx.y][threadIdx.x + 1] += 1;
   }

   __syncthreads();

   if(threadIdx.x == 0) {
      okToGo[threadIdx.y][threadIdx.x] = 1;
   }

   while(!okToGo[threadIdx.y][threadIdx.x]) {};

   tmp = result + carry[threadIdx.y][threadIdx.x];
   if(tmp < result && threadIdx.x < BLOCKDIM - 1) {
      carry[threadIdx.y][threadIdx.x] += 1;
   }
   
   if(threadIdx < BLOCKDIM - 1) {
      okayToGo[threadIdx.y][threadIdx.x] = 1;
   }

   res[threadIdx.x] = tmp;

   __syncthreads();*/
}

__device__ bool geq(BigInt *n, BigInt *m) {

   __shared__ int differPos[BLOCKDIM_Y];
   int *pos = differPos + threadIdx.y;

   if(threadIdx.x == 0) {
      differPos[threadIdx.y] = 0;
   }

   if(n->components[threadIdx.x] != m->components[threadIdx.x]) {
      atomicMax(pos, threadIdx.x);
   }

   return n->components[*pos] >= m->components[*pos];
}


   /*keep a shared mem array that contains flags for thread actions
   __shared__ short gThan[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   short res;

   /*Set the Most significant thread to a val to start the process
   if(threadIdx.x == BLOCKDIM_X - 1) {
      gThan[threadIdx.y][threadIdx.x] = NOTSURE;
   }

   /*Loop the threads until a flag is set for them
   while(!gThan[threadIdx.y][threadIdx.x]){};

   res = gThan[threadIdx.y][threadIdx.x];

   /* If a more significant thread found out that the numbers
    * were greater, notify the less significant threads and
    * return.
   if(res == ISGREATER) {
      /* The least significant thread doesn't need to
       * notify anyone.
      if(threadIdx.x != 0) {
         gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
      }

      return true;
   }

   /* If a more significant thread found out that the numbers
    * were not greater, notify the less significant threads and
    * return.
   if(res == NOTGREATER) {
      /* The least significant thread doesn't need to
       * notify anyone.
      if(threadIdx.x != 0) {
         gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
      }

      return true;
   }

   /* If a more significant thread was unsure whether the
    * numbers were greater than or not, check the numbers
    * handled by THIS current thread.
   if(res == NOTSURE) {
      /* If the numbers handled by this thread show n is greater
       * than m, then notify the more significant threads, and
       * the lesser significant threads, and return.
      if(n->components[threadIdx.x] > m->components[threadIdx.x]) {
         if(threadIdx.x != BLOCKDIM_X - 1) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
         }
         if(threadIdx.x != 0) {
            gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
         }
         return true;
      }

      /* If the numbers handled by this thread show n is not greater
       * than m, then notify the more significant threads, and
       * the lesser significant threads, and return.
      else if(n->components[threadIdx.x] < m->components[threadIdx.x]) {
         if(threadIdx.x != BLOCKDIM_X - 1) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_NOT_GREATER;
         }
         if(threadIdx.x != 0) {
            gThan[threadIdx.y][threadIdx.x - 1] = NOTGREATER;
         }
         return false;
      }

      /* If the numbers handled by this thread are equal ...
      else {
         /* If this thread is the least significant thread, 
          * notify all other threads and return.
         if(threadIdx.x == 0) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
            return true;
         }
         
         /* If the number is not the least significant thread,
          * tell the next less significant thread it is unsure.
         gThan[threadIdx.y][threadIdx.x - 1] = NOTSURE;

      }
   } 

   res = gThan[threadIdx.y][threadIdx.x];
   
   /* Wait for the lesser significant threads to notify this
    * thread of an answer if it was unsure.
   while(res != NOTIFY_IS_GREATER || res != NOTIFY_NOT_GREATER) {
      res = gThan[threadIdx.y][threadIdx.x];
   }

   if(res == NOTIFY_IS_GREATER) {
      gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
      return true;
   }
   
   gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_NOT_GREATER;
   return false;*/
 
__device__ bool notOne(BigInt n) {
   __shared__ bool notOne[BLOCKDIM] = {{false}};
   int ind = threadIdx.x;
   int key = threadIdx.y;

   if(ind != 0 && n.components[ind] != 0) {
      notOne[key] = true;
   }
   if(ind == 0 && n.components[ind] != 1) {
      notOne[key] = true;
   }

   return notOne[key];
}

__device__ bool notZero(BigInt n) {
   __shared__ bool isZero[BLOCKDIM] = {{true}};
   int ind = threadIdx.x;

   if(__any(threadIdx.x)) {
      return false;
   }
   /*if(n.components[ind] != 0) {
      isZero[threadIdx.y] = false;
   }
   __syncthreads();

   return isZero[threadIdx.y];*/
}

__device__ BigInt* gcd(BigInt *n, BigInt *m) {

   int i;
   unsigned int tmp;
   BigInt *tmpptr;

   for(i = 0; ((n->components[0] | m->components[0]) & 1) == 0; i++) {
      shiftR(n);
      shiftR(m);
   }

   tmp = n->components[0];

   while ((tmp & 1) == 0) {
      shiftR(n);
      tmp = n->components[0];
   }

   do {
      while((m->components[0] & 1) == 0) {
         shiftR(m);
      }

      if(geq(n, m)) {
         tmpptr = n->components;
         n->components= m->components;
         m->components = tmpptr;
      }

      cuSub(m, n);
   } while(notZero(*n));
   while(i != 0) {
      shiftL(m);
      i--;
   }
   
   return m;
}

