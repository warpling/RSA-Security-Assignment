#include "hip/hip_runtime.h"
#include "cudaFunctions.h"

__global__ void gcdKernel(int ind1, int ind2, int totNumKeys, BigInt keys[], int res) {
   
   int row = blockIdx.y*blockDim.y + threadIdx.y;
   int b, i;
   __shared__ int results[BLOCKDIM_Y][BLOCKDIM_X];
   BigInt n = keys[ind1+row];
   BigInt tmp = n;
   BigInt m; 

   for(i = ind1+row + 1; i < totNumKeys; i++) {
      m = keys[i];
      m = gcd(tmp, m);
      if(notOne(m)) {
         res[totNumKeys*(ind1+row) + i] = 1;
      }

   }
}

__device__ void  shiftR(BigInt *n) {
   unsigned int part = 0;
   int id = threadIdx.x;

   if(id != 31) {
      part = n.elements[threadIdx.x + 1];
   }
   n.elements[threadIdx.x] = (n.elements[threadIdx.x] >> 1) | (part << 31);
   
   /*unsigned int part = n->elements[threadIdx.x];
   __shared__ unsigned int rsResult[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   unsigned int mask = 1;

   if(threadIdx.x != 0) {
      rsResult[threadIdx.y][threadIdx.x - 1] |= (part & mask) << 31;
   }

   part >>= 1;

   if(threadIdx.x != blockDim.x - 1) {
      rsResult[threadIdx.y][threadIdx.x] |= part;
   }
   __syncThreads();
   n->elements[threadIdx.x] = rsResult[threadIdx.y][threadIdx.x];
   __syncthreads();*/
}

__device__ void  shiftL(BigInt *n) {
   unsigned int part = 0;

   if(ThreadIdx.x) {
      part = n.elements[threadIdx.x - 1];
   }
   n.elements[threadIdx.x] = (n.elements[threadIdx.x] << 1) | (part >> 31);

   /*unsigned int part = n->elements[threadIdx.x];
   __shared__ unsigned int lOverflow[BLOCKDIM_Y][BLOCKDIM_X];
   unsigned int mask = 1 << 31;

   if(threadIdx.x != blockDim.x - 1) {
      lOverflow.elements[threadIdx.y][threadIdx.x - 1] = (part & mask) >> 31;
   }

   part <<= 1;
   __syncthreads();

   if(threadIdx.x != 0)
      part = part | lOverflow.elements[threadIdx.y][threadIdx.x];
   }

   n->elements[threadIdx.x] = part;
   __syncthreads();*/
}

__device__ void cuSubtract(BigInt *n, BigInt *m, unsigned int *res) {

   unsigned int partn = n->elements[threadIdx.x];
   unsigned int partm = m->elements[threadIdx.x];
   __shared__ unsigned int borrowArray[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   //__shared__ short okayToGo[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   //unsigned int result = 0;
   unsigned int tmp;

   //partm = ~partm;

   /*if(threadIdx.x == 31) {
      borrowArray[threadIdx.y][0] = 0;
   }*/

   tmp = partn - partm;
   if(threadIdx != 0 && tmp > partn) {
      borrowArray[threadIdx.y][threadIdx.x + 1] = 1;
   }
   else {
      borrowArray[threadIdx.y][threadIdx.x] = 0;
   }

   while(__any(borrowArray[threadIdx.y][threadIdx.x])) {
      if(borrowArray[threadIdx.y][threadIdx.x]) {
         tmp --;
      }
      if(threadIdx.x != 31 && t == 0xffffffffU) {
         borrowArray[threadIdx.y][threadIdx.x + 1] = 1;
      }
   }
   n->elements[threadIdx.x] = tmp;



      /*tmp = partm++;
      if(tmp < partm) {
         carry[threadIdx.y][threadIdx.x + 1] += 1;
      }
      partm = tmp;*/
   /*}

   res = partn + partm;
   if((res < partn || res < partm) && threadIdx.x < BLOCKDIM - 1) {
      carry[threadIdx.y][threadIdx.x + 1] += 1;
   }

   __syncthreads();

   if(threadIdx.x == 0) {
      okToGo[threadIdx.y][threadIdx.x] = 1;
   }

   while(!okToGo[threadIdx.y][threadIdx.x]) {};

   tmp = result + carry[threadIdx.y][threadIdx.x];
   if(tmp < result && threadIdx.x < BLOCKDIM - 1) {
      carry[threadIdx.y][threadIdx.x] += 1;
   }
   
   if(threadIdx < BLOCKDIM - 1) {
      okayToGo[threadIdx.y][threadIdx.x] = 1;
   }

   res[threadIdx.x] = tmp;

   __syncthreads();*/
}

__device__ bool geq(BigInt *n, BigInt *m) {

   /*keep a shared mem array that contains flags for thread actions*/
   __shared__ short gThan[BLOCKDIM_Y][BLOCKDIM_X] = {{0}};
   short res;

   /*Set the Most significant thread to a val to start the process*/
   if(threadIdx.x == BLOCKDIM_X - 1) {
      gThan[threadIdx.y][threadIdx.x] = NOTSURE;
   }

   /*Loop the threads until a flag is set for them*/
   while(!gThan[threadIdx.y][threadIdx.x]){};

   res = gThan[threadIdx.y][threadIdx.x];

   /* If a more significant thread found out that the numbers
    * were greater, notify the less significant threads and
    * return.
    */
   if(res == ISGREATER) {
      /* The least significant thread doesn't need to
       * notify anyone.
       */
      if(threadIdx.x != 0) {
         gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
      }

      return true;
   }

   /* If a more significant thread found out that the numbers
    * were not greater, notify the less significant threads and
    * return.
    */
   if(res == NOTGREATER) {
      /* The least significant thread doesn't need to
       * notify anyone.
       */
      if(threadIdx.x != 0) {
         gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
      }

      return true;
   }

   /* If a more significant thread was unsure whether the
    * numbers were greater than or not, check the numbers
    * handled by THIS current thread.
    */
   if(res == NOTSURE) {
      /* If the numbers handled by this thread show n is greater
       * than m, then notify the more significant threads, and
       * the lesser significant threads, and return.
       */
      if(n->elements[threadIdx.x] > m->elements[threadIdx.x]) {
         if(threadIdx.x != BLOCKDIM_X - 1) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
         }
         if(threadIdx.x != 0) {
            gThan[threadIdx.y][threadIdx.x - 1] = ISGREATER;
         }
         return true;
      }

      /* If the numbers handled by this thread show n is not greater
       * than m, then notify the more significant threads, and
       * the lesser significant threads, and return.
       */
      else if(n->elements[threadIdx.x] < m->elements[threadIdx.x]) {
         if(threadIdx.x != BLOCKDIM_X - 1) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_NOT_GREATER;
         }
         if(threadIdx.x != 0) {
            gThan[threadIdx.y][threadIdx.x - 1] = NOTGREATER;
         }
         return false;
      }

      /* If the numbers handled by this thread are equal ...*/
      else {
         /* If this thread is the least significant thread, 
          * notify all other threads and return.
          */
         if(threadIdx.x == 0) {
            gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
            return true;
         }
         
         /* If the number is not the least significant thread,
          * tell the next less significant thread it is unsure.
          */
         gThan[threadIdx.y][threadIdx.x - 1] = NOTSURE;

      }
   } 

   res = gThan[threadIdx.y][threadIdx.x];
   
   /* Wait for the lesser significant threads to notify this
    * thread of an answer if it was unsure.
    */
   while(res != NOTIFY_IS_GREATER || res != NOTIFY_NOT_GREATER) {
      res = gThan[threadIdx.y][threadIdx.x];
   }

   if(res == NOTIFY_IS_GREATER) {
      gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_IS_GREATER;
      return true;
   }
   
   gThan[threadIdx.y][threadIdx.x + 1] = NOTIFY_NOT_GREATER;
   return false;
}
 
__device__ bool notOne(BigInt n) {
   __shared__ bool notOne[BLOCKDIM] = {{false}};
   int ind = threadIdx.x;
   int key = threadIdx.y;

   if(ind != 0 && n.elements[ind] != 0) {
      notOne[key] = true;
   }
   if(ind == 0 && n.elements[ind] != 1) {
      notOne[key] = true;
   __syncthreads();

   return notOne[key];
}

__device__ bool notZero(BigInt n) {
   __shared__ bool isZero[BLOCKDIM] = {{true}};
   int ind = threadIdx.x;

   if(n.elements[ind] != 0) {
      isZero[threadIdx.y] = false;
   }
   __syncthreads();

   return isZero[threadIdx.y];
}

__device__ BigInt gcd(BigInt n, BigInt m) {

   int i;
   unsigned int tmp;

   for(i = 0; ((n.elements[0] | m.elements[0]) & 1) == 0; i++) {
      shiftR(&n);
      shiftR(&m);
   }

   tmp = n.elements[0];

   while ((tmp & 1) == 0) {
      shiftR(&n);
      tmp = n.elements[0];
   }

   do {
      while((m.elements[0] & 1) == 0) {
         shiftR(&m);
      }

      if(geq(&n, &m)) {
         tmp = n.elements[threadIdx.x];
         n.elements[threadIdx.x] = m.elements[threadIdx.x];
         m.elements[threadIdx.x] = tmp;
      }
      __syncthreads();

      cuSub(n, m);
   } while(notZero(n));
   while(i != 0) {
      shiftL(&m);
      i--;
   }
   
   return m;
}

















