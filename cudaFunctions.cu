#include "hip/hip_runtime.h"
#include "bigInt.h"
#include "cudaFunctions.h"

/*__global__ void subTest(bigInt *key1, bigInt *key2, bigInt *result) {
   int i;
   uint32_t *ptr;
   uint32_t *keyptr1, *keyptr2;
   __shared__ uint32_t keyComp1[BLOCKDIM_X];
   __shared__ uint32_t keyComp2[BLOCKDIM_X];
   keyComp1[threadIdx.x] = key1->components[threadIdx.x];
   keyComp2[threadIdx.x] = key2->components[threadIdx.x];
   keyptr1 = keyComp1;
   keyptr2 = keyComp2;
   printf("here!\n");

   ptr = gcd(keyptr1, keyptr2);
   result->components[threadIdx.x] = ptr[threadIdx.x];
   if(threadIdx.x == 0) {
      printf("gcd = %d\n", result->components[threadIdx.x]);
   }
   //for(i = 0; i < 156; i ++) {
      //shiftL(key1.components);
   //}
   
   //cuSubtract(key1.components, key2.components);
   /*if(geq(&key1, &key2)) {
      result.components[threadIdx.x] = 1;
   }
   else {
      result.components[threadIdx.x] = 0;
   }
   //result.components[threadIdx.x] = key1.components[threadIdx.x];
} */

__global__ void gcdKernel(int base, int offset, bigInt *keys, int numKeys, uint32_t *results) {
   
   int key1 = base;
   int key2 = offset + blockIdx.x;
   uint32_t mask = (uint32_t)(1 << 31);
   uint32_t *keyptr1, *keyptr2;
   __shared__ uint32_t sharedkey1[BLOCKDIM_X];
   __shared__ uint32_t sharedkey2[BLOCKDIM_X];
   uint32_t *res;
   
   
   if(key1 < key2 && key2 < numKeys && key1 < numKeys) {
      sharedkey1[threadIdx.x] = keys[key1].components[threadIdx.x];
      sharedkey2[threadIdx.x] = keys[key2].components[threadIdx.x];
      __syncthreads();
      keyptr1 = sharedkey1;
      keyptr2 = sharedkey2;
      res = gcd(keyptr1, keyptr2);
      if(notOne(res)) {
         atomicOr(results + (key1/32), mask >> (key1%32));
         atomicOr(results + (key2/32), mask >> (key2%32));
         /*if(threadIdx.x == 0) {
            //printf("found bad key!, gcd = %d, key1: %d, key2: %d\n", res[threadIdx.x], key1, key2);
         }*/
      }
   }
}

__device__ void  shiftR(uint32_t *n) {
   uint32_t part = 0;
   uint32_t tmp;
   int id = threadIdx.x;

   if(id != 31) {
      part = n[threadIdx.x + 1];
   }
   tmp = (n[threadIdx.x] >> 1) | (part << 31);
   n[threadIdx.x] = tmp;
}

__device__ void  shiftL(uint32_t *n) {
   uint32_t part = 0;

   if(threadIdx.x) {
      part = n[threadIdx.x - 1];
   }
   n[threadIdx.x] = (n[threadIdx.x] << 1) | (part >> 31);
}

__device__ void cuSubtract(uint32_t *n, uint32_t *m) {
   uint32_t partn = n[threadIdx.x];
   uint32_t partm = m[threadIdx.x];
   uint32_t tmp;
   int carry;

   n[threadIdx.x] = partn - partm;
   tmp = n[threadIdx.x];
   if(threadIdx.x != 31) {
      carry = tmp > partn;
   }
   partn = tmp;
   
   while(__any(carry)) {
      if(threadIdx.x == 31) {
         carry = 0;
      }
      else if(carry) {
         n[threadIdx.x + 1] --;
         carry = 0;
      }
      if(n[threadIdx.x] > tmp && threadIdx.x != 31) {
         carry = 1;
      }
      tmp = n[threadIdx.x];
   }
   n[threadIdx.x] = tmp;
}

__device__ bool geq(uint32_t *n, uint32_t *m) {

   __shared__ int differPos[BLOCKDIM_Y];
   int *pos = differPos + threadIdx.y;

   if(threadIdx.x == 0) {
      differPos[threadIdx.y] = 0;
   }

   if(n[threadIdx.x] != m[threadIdx.x]) {
      atomicMax(pos, threadIdx.x);
   }

   return n[*pos] >= m[*pos];
}
 
__device__ bool notOne(uint32_t *n) {
   int ind = threadIdx.x;
   int res = 0;

   if(ind != 0 && n[ind] != 0) {
      res = 1;
      
   }
   if(ind == 0 && n[ind] != 1) {
      res = 1;
   }
   if(__any(res)) {
      res = 1;
   }

   return res;
}

__device__ bool notZero(uint32_t *n) {

   if(__any(n[threadIdx.x])) {
      return true;
   }
   return false;
}

__device__ uint32_t* gcd(uint32_t *n, uint32_t *m) {

   int i;
   uint32_t tmp;
   uint32_t *tmpptr;

   for(i = 0; ((n[0] | m[0]) & 1) == 0; i++) {
      shiftR(n);
      shiftR(m);
   }

   tmp = n[0];

   while ((tmp & 1) == 0) {
      shiftR(n);
      tmp = n[0];
   }

   do {
      while((m[0] & 1) == 0) {
         shiftR(m);
      }

      if(geq(n, m)) {
         tmpptr = n;
         n = m;
         m = tmpptr;
      }

      cuSubtract(m, n);
   } while(notZero(m));
   while(i != 0) {
      shiftL(n);
      i--;
   }
   return n;
}

